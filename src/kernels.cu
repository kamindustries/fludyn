#include "hip/hip_runtime.h"
#include "kernels.cuh"


__device__ int
clamp(int i)
{
	if (i < 0) i = 0;
	if (i > 255) i = 255;
	return i;
}

__device__ float
clamp(float i, float min, float max)
{
	if (i < min) i = min;
	if (i > max) i = max;
	return i;
}

__device__ float
fitRange(float valueIn, float baseMin, float baseMax, float limitMin, float limitMax)
{
	return ((limitMax - limitMin) * (valueIn - baseMin) / (baseMax - baseMin)) + limitMin;
}

// Get 1d index from 2d coords
__device__ int
IX(int x, int y)
{
	return x + (y * blockDim.x * gridDim.x);
}

__device__ int
getX(int w)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	//if (x >= w) x = 0; if (x < 0) x = w-1;
	return x;
}

__device__ int
getY(int h)
{
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	//if (y >= h) y = 0; if (y < 0) y = h-1;
	return y;
}


// Returns true if within the bounds of both the container edges and a user-defined boundary
__device__ bool
checkBounds(int *_boundary, int x, int y, int w, int h)
{
	if (x > 1 && x < w-2 && y > 1 && y < h-2 && _boundary[IX(x,y)] < 1 ){
		return true;
	}
	else {
		return false;
	}
}
__device__ bool
checkBounds(int x, int y, int w, int h)
{
	if (x > 1 && x < w-2 && y > 1 && y < h-2){
		return true;
	}
	else {
		return false;
	}
}

// Functions for converting to/from a int (4 bytes, 1 byte per RGBA, which are in the range 0-255)
// to 4 floats in the range 0.0-1.0
// Note how the data is stored in BGRA format due to how its stored on the GPU.
__device__ int
rgbaToInt(float r, float g, float b, float a)
{
    return
		(clamp((int)(a * 255.0f)) << 24) |
		(clamp((int)(r * 255.0f)) << 16) |
		(clamp((int)(g * 255.0f)) <<  8) |
		(clamp((int)(b * 255.0f)) <<  0);
}

__device__ void
intToRgba(int pixel, float &r, float &g, float &b, float &a)
{
	b = float(pixel&0xff) / 255.0f;
	g = float((pixel>>8)&0xff) / 255.0f;
	r = float((pixel>>16)&0xff) / 255.0f;
	a = float((pixel>>24)&0xff) / 255.0f;
}

__device__ void
rgbaToColor(float *dest, int id, float r, float g, float b, float a)
{
	dest[4*id]=b;
	dest[4*id+1] = g;
	dest[4*id+2] = r;
	dest[4*id+3] = a;

}

// Set boundary conditions
__device__ void set_bnd( int b, int x, int y, float *field, int *boundary, int w, int h) {
	int sz = w*h;
	int id = IX(x,y);

	bool outOfBnd = false;
	if (boundary[id] > 0) outOfBnd = true;

	//if (x==0)	field[id] = b==1 ? -1*field[IX(1,y)] : field[IX(1,y)];
	//if (x==w-1) field[id] = b==1 ? -1*field[IX(w-2,y)] : field[IX(w-2,y)];
	//if (y==0)   field[id] = b==2 ? -1*field[IX(x,1)] : field[IX(x,1)];
	//if (y==h-1) field[id] = b==2 ? -1*field[IX(x,h-2)] : field[IX(x,h-2)];

	if (x==0)	field[id] = b==1 ? -1*field[IX(1,y)] : -1 * field[IX(1,y)];
	if (x==w-1) field[id] = b==1 ? -1*field[IX(w-2,y)] : -1 * field[IX(w-2,y)];
	if (y==0)   field[id] = b==2 ? -1*field[IX(x,1)] : -1 * field[IX(x,1)];
	if (y==h-1) field[id] = b==2 ? -1*field[IX(x,h-2)] : -1 * field[IX(x,h-2)];

	//if (outOfBnd){
	//	field[id] = -1*field[id];
	//	field[IX(x+1,y)] = -1*field[IX(x+1,y)];
	//	field[IX(x-1,y)] = -1*field[IX(x-1,y)];
	//	field[IX(x,y+1)] = -1*field[IX(x,y+1)];
	//	field[IX(x,y-1)] = -1*field[IX(x,y-1)];
	//}

	if (id == 0)      field[id] = 0.5*(field[IX(1,0)]+field[IX(0,1)]);  // southwest
	if (id == sz-w) field[id] = 0.5*(field[IX(1,h-1)]+field[IX(0, h-2)]); // northwest
	if (id == w-1)  field[id] = 0.5*(field[IX(w-2,0)]+field[IX(w-1,1)]); // southeast
	if (id == sz-1)   field[id] = 0.5*(field[IX(w-2,h-1)]+field[IX(w-1,h-2)]); // northeast
}

__global__ void
DrawSquare( float *field, float value, int w, int h ) {
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	float posX = (float)x/w;
	float posY = (float)y/h;
	if ( posX < .92 && posX > .45 && posY < .51 && posY > .495 ) {
		field[id] = value;
	}
}

__global__ void
DrawBnd( int *boundary, int w, int h ) {
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	float posX = (float)x/w;
	float posY = (float)y/h;
	if ( posX < .82 && posX > .70 && posY < .33 && posY > .21 ) {
		boundary[id] = 1;
	}
	else boundary[id] = 0;
}

__global__ void SetBoundary( int b, float *field, int *boundary, int w, int h ) {
	int x = getX(w);
	int y = getY(h);

	set_bnd(b, x, y, field, boundary, w, h);
}

__global__ void getSum( float *_data, float _sum, int w, int h ) {
  int x = getX(w);
  int y = getY(h);

  _sum += _data[IX(x,y)];
}

__global__ void
ClearArray(float *field, float value, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	field[id] = value;
}

__global__ void
ClearArray(int *field, float value, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	field[id] = value;
}

__global__ void GetFromUI ( float * field, float value, int x_coord, int y_coord, int w, int h ) {
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x>x_coord-5 && x<x_coord+5 && y>y_coord-5 && y<y_coord+5){
		// if (x == x_coord && y==y_coord){
		field[id] += value;
	}
	else return;
}

__global__ void
MakeSource(int *src, float *dest, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	int pixel = src[id];
	float r,g,b,a;
	intToRgba(pixel, r, g, b, a);

	dest[id] = r;
}

// *!* This is currently only grabbing the red channel *!*
__global__ void
MakeSource(int *src, int *dest, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	int pixel = src[id];
	float r,g,b,a;
	intToRgba(pixel, r, g, b, a);

	dest[id] = src[id]&0xff/255;
}

__global__ void
AddSource(float *field, float *source, float dt, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	field[id] += (dt * source[id]);
}

__global__ void
MakeColor(float *src, int *dest, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	dest[id] = rgbaToInt(src[id], src[id], src[id], 1.0);
	//dest[id] = rgbaToInt(1.0, src[id], src[id], 1.0);
}

__global__ void
MakeColor(float *src0, float *src1, float *src2, float *dest, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	rgbaToColor(dest, id, src0[id], src1[id], src2[id], 1.0);
}


__global__ void
MakeColor(float *src0, float *src1, float *src2, float *src3, float *dest, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	rgbaToColor(dest, id, src0[id], src1[id], src2[id], src3[id]);
}

__global__ void
MakeColor( float *data, float4 *toDisplay, int w, int h) {
  int x = getX(w);
  int y = getY(h);
  int id = IX(x,y);

  float Cd = data[id];
  toDisplay[id] = make_float4(Cd, Cd, Cd, 1.0);
}

__global__ void
TEST (float *test, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	//test[0][id] = .5;
	//test[1][id] = .5;
	test[id] = .5;
}

__device__ float
bilerp(float *src, float i, float j, int w, int h)
{
	int i0, j0, i1, j1;
	float s0, t0, s1, t1;

	// fit bounds
	if (i < 0.5f) i = 0.5f;
	if (i > float(w)-2.0+0.5f) i = float(w)-2.0+0.5f;
	if (j < 0.5f) j = 0.5f;
	if (j > float(h)-2.0+0.5f) j = float(h)-2.0+0.5f;

	// bilinear interpolation
	i0 = int(i);
	i1 = i0+1;
	j0 = int(j);
	j1 = j0+1;

	s1 = (float)i-i0;
	s0 = (float)1-s1;
	t1 = (float)j-j0;
	t0 = (float)1-t1;

	return (float)	s0*(t0*src[IX(i0,j0)] + t1*src[IX(i0,j1)])+
			 		s1*(t0*src[IX(i1,j0)] + t1*src[IX(i1,j1)]);
}

__global__ void Advect (float *vel_u, float *vel_v, float *src_u, float *src_v,
						int *boundary, float *dest_u, float *dest_v,
						float timeStep, float diff, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x > 1 && x < w-1 && y > 1 && y < h-1){
		float dt0 = (float)timeStep * float(w-2);
		float i = float(x) - dt0 * vel_u[id];
		float j = float(y) - dt0 * vel_v[id];

		dest_u[id] = diff * bilerp(src_u, i, j, w, h);
		dest_v[id] = diff * bilerp(src_v, i, j, w, h);
	}

	if (!checkBounds(boundary, x, y, w, h)) {
		dest_u[id] = 0.0;
		dest_v[id] = 0.0;
	}

}

__global__ void Advect (float *vel_u, float *vel_v, float *src, int *boundary, float *dest,
						float timeStep, float diff, bool skipBilerp, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x > 1 && x < w-1 && y > 1 && y < h-1){
		float dt0 = (float)timeStep * float(w-2);
		float i = float(x) - dt0 * vel_u[id];
		float j = float(y) - dt0 * vel_v[id];

		dest[id] = diff * bilerp(src, i, j, w, h);

		//if (skipBilerp) {
		//	int c_x = x - timeStep * vel_u[id];
		//	int c_y = y - timeStep * vel_v[id];
		//	dest[id] = src[IX(c_x, c_y)];
		//}
	}

	if (!checkBounds(boundary, x, y, w, h)) {
		dest[id] = 0.0;
	}
}

__device__ float curl(int i, int j, float *u, float *v)
{
	float du_dy = (u[IX(i, j+1)] - u[IX(i, j-1)]) * 0.5f;
	float dv_dx = (v[IX(i+1, j)] - v[IX(i-1, j)]) * 0.5f;

	return du_dy - dv_dx;
}

__global__ void vorticityConfinement(float *u, float *v, float *Fvc_x, float *Fvc_y, int *_boundary,
								     float dt, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	float dw_dx, dw_dy;
	float length;
	float vel;

	//if (x>1 && x<w-2 && y>1 && y<h-2){
	if (checkBounds(_boundary, x, y, w, h)) {

		// Calculate magnitude of curl(u,v) for each cell. (|w|)
		// curl[I(i, j)] = Math.abs(curl(i, j));

		// Find derivative of the magnitude (n = del |w|)
		dw_dx = ( abs(curl(x+1,y, u, v)) - abs(curl(x-1,y, u, v)) ) * 0.5f;
		dw_dy = ( abs(curl(x,y+1, u, v)) - abs(curl(x,y-1, u, v)) ) * 0.5f;

		// Calculate vector length. (|n|)
		// Add small factor to prevent divide by zeros.
		length = sqrt(dw_dx * dw_dx + dw_dy * dw_dy);
		if (length == 0.0) length -= 0.000001f;
		// N = ( n/|n| )
		dw_dx /= length;
		dw_dy /= length;

		vel = curl(x, y, u, v);

		// N x w
		// 0.5 = curl amount
		Fvc_x[id] += (dw_dy * -vel * dt * 0.5);
		Fvc_y[id] += (dw_dx *  vel * dt * 0.5);
	}
}


__global__ void ApplyBuoyancy( float *vel_u, float *vel_v, float *temp, float *dens,
							   float *dest_u, float *dest_v, float ambientTemp, float dt, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (checkBounds(x, y, w, h)) {
		dest_u[id] = vel_u[id];
		dest_v[id] = vel_v[id];

		float T = temp[id];
		float Sigma = 1.0;
		float Kappa = 0.05;
		if (T > ambientTemp) {
			float D = dens[id];
			float dt0 = (float)dt;

			dest_u[id] += (dt0 * (T - ambientTemp) * Sigma - D * Kappa) * 0;
			dest_v[id] += (dt0 * (T - ambientTemp) * Sigma - D * Kappa) * .01;
		}
	}

}

__global__ void ComputeDivergence( float *u, float *v, int *boundary, float *dest, int w, int h )
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x > 2 && x < w-2 && y > 2 && y < h-2){
		float cellSize = 1.0;
		//dest[id] = (0.5 / cellSize) * ( u[IX(x+1, y)] - u[IX(x-1, y)] + v[IX(x, y+1)] - v[IX(x, y-1)] );
		//dest[id] = 0.5 * ( (u[IX(x+1, y)] - u[IX(x-1, y)]) + (v[IX(x, y+1)] - v[IX(x, y-1)]) ) ;
		dest[id] = 0.5 * ( u[IX(x+1, y)] - u[IX(x-1, y)] + v[IX(x, y+1)] - v[IX(x, y-1)] ) / float(w-2);
	}
}

__global__ void Jacobi( float *p, float *divergence, int *boundary, float *dest, int w, int h )
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x > 1 && x < w-1 && y > 1 && y < h-1){
		// Find neighboring pressure:
		float pN = p[IX(x, y+1)];
		float pS = p[IX(x, y-1)];
		float pE = p[IX(x+1, y)];
		float pW = p[IX(x-1, y)];
		float pC = p[id];

		// Find neighboring obstacles:
		int oN = boundary[IX(x, y+1)];
		int oS = boundary[IX(x, y-1)];
		int oE = boundary[IX(x+1, y)];
		int oW = boundary[IX(x-1, y)];

		// Use center pressure for solid cells:
		if (oN > 0) pN = pC;
		if (oS > 0) pS = pC;
		if (oE > 0) pE = pC;
		if (oW > 0) pW = pC;

		float cellSize = 1.0;
		//float Alpha = -cellSize * cellSize;
		float Alpha = -1.0;
		float bC = divergence[id];
		float InverseBeta = .25;
		dest[id] = (pW + pE + pS + pN + Alpha * bC) * InverseBeta;
		//dest[id] = (divergence[id] + (1.0*(p[IX(x+1,y)] + p[IX(x-1,y)] + p[IX(x,y+1)] + p[IX(x,y-1)]))) * .25;
	}
}

__global__ void SubtractGradient( float *vel_u, float *vel_v, float *p, int *boundary,
								  float *dest_u, float *dest_v, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (x > 1 && x < w-1 && y > 1 && y < h-1){
		// Find neighboring pressure:
		float pN = p[IX(x, y+1)];
		float pS = p[IX(x, y-1)];
		float pE = p[IX(x+1, y)];
		float pW = p[IX(x-1, y)];
		float pC = p[id];

		// Find neighboring obstacles:
		int oN = boundary[IX(x, y+1)];
		int oS = boundary[IX(x, y-1)];
		int oE = boundary[IX(x+1, y)];
		int oW = boundary[IX(x-1, y)];

		// Use center pressure for solid cells:
		float obstV = 0.0;
		float vMask = 1.0;

		if (oN > 0) { pN = pC; vMask = 0.0; }
		if (oS > 0) {pS = pC; vMask = 0.0; }
		if (oE > 0) {pE = pC; vMask = 0.0; }
		if (oW > 0) {pW = pC; vMask = 0.0; }

		// Enforce the free-slip boundary condition:
		float old_u = vel_u[id];
		float old_v = vel_v[id];

		// GradientScale is 1.125 / CellSize
		float cellSize = 1.0;
		//float GradientScale = 1.125 / cellSize;
		float GradientScale = 0.5 * float(w-2);
		float grad_u = (pE - pW) * GradientScale;
		float grad_v = (pN - pS) * GradientScale;

		float new_u = old_u - grad_u;
		float new_v = old_v - grad_v;

		dest_u[id] = (vMask * new_u) + obstV;
		dest_v[id] = (vMask * new_v) + obstV;

	}
}


__global__ void
Diffusion(float *_chem, float *_lap, int *_boundary, float _difConst, float dt, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	// have to do this check for non-powers of 2 to work...?
	if (checkBounds(_boundary, x, y, w, h)) {

		// constants
		float xLength = (float)x/100.0;
		float dx = (float)xLength/(float)x;
		float alpha = (float)(_difConst * dt / (float)(dx*dx));

		int n1 = getX(x-1);
		int n2 = getX(x+1);
		int n3 = getY(y-1);
		int n4 = getY(y+1);
		_lap[id] = (float)(-4.0f * _chem[id]) + (float)(_chem[IX(x+1,y)] + _chem[IX(x-1,y)] + _chem[IX(x,y+1)] + _chem[IX(x,y-1)]);
		_lap[id] = (float)_lap[id]*alpha;
	}
}

__global__ void
AddLaplacian( float *_chem, float *_lap, int w, int h)
{
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	_chem[id] += _lap[id];
}

__global__ void React( float *_chemA, float *_chemB, float *F_input, float *rd, int *_boundary, float dt, int w, int h) {
	int x = getX(w);
	int y = getY(h);
	int id = IX(x,y);

	if (checkBounds(_boundary, x, y, w, h)) {
		//float F = 0.05;
		//float k = 0.0675;
		//float F = 0.0140;
		//float k = 0.0490;
		//float F = 0.0545;
		//float k = 0.062;
		//float F = F_input[id]&0xff/255;
		//F = fitRange(F, 0.0, 1.0, 0.014, 0.066);
		//
		//float k = 1.0 - (F_input[id]&0xff/255);
		//k = fitRange(k, 0.0, 1.0, 0.05, 0.068);
		float F = rd[0];
		float k = rd[1];

		float A = _chemA[id];
		float B = _chemB[id];

		float reactionA = -A * (B*B) + (F * (1.0-A));
		float reactionB = A * (B*B) - (F+k)*B;
		_chemA[id] += (dt * reactionA);
		_chemB[id] += (dt * reactionB);
	}
	else {
		_chemA[id] *= -1.0;
		_chemB[id] *= -1.0;
	}
}
