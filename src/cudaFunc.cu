#include "hip/hip_runtime.h"
#include "cudaFunc.cuh"

void initCUDA(hipGraphicsResource_t cgrTx, GLuint txBuffer) {
  // checkCudaErrors( hipSetDevice(gpuGetMaxGflopsDeviceId()) );
  // checkCudaErrors( cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId()) );
  hipGraphicsGLRegisterBuffer( &cgrTx, txBuffer, cudaGraphicsMapFlagsWriteDiscard );
  // hipGraphicsGLRegisterBuffer( &cgrVertData, vertexArrayID, cudaGraphicsMapFlagsWriteDiscard );

  hipMalloc((void**)&chemA, sizeof(float)*size);
	hipMalloc((void**)&chemA_prev, sizeof(float)*size);
	hipMalloc((void**)&chemB, sizeof(float)*size);
	hipMalloc((void**)&chemB_prev, sizeof(float)*size);
	hipMalloc((void**)&laplacian, sizeof(float)*size);
	hipMalloc((void**)&boundary, sizeof(int)*size);

	for (int i=0; i<2; i++){
		hipMalloc((void**)&vel[i], sizeof(int)*size);
		hipMalloc((void**)&vel_prev[i], sizeof(int)*size);
	}

	hipMalloc((void**)&pressure, sizeof(float)*size );
	hipMalloc((void**)&pressure_prev, sizeof(float)*size );
	hipMalloc((void**)&temperature, sizeof(float)*size );
	hipMalloc((void**)&temperature_prev, sizeof(float)*size );
	hipMalloc((void**)&density, sizeof(float)*size );
	hipMalloc((void**)&density_prev, sizeof(float)*size );
	hipMalloc((void**)&divergence, sizeof(float)*size );
}

void initGPUArrays() {
  for (int i=0; i<2; i++){
	  ClearArray<<<grid,threads>>>(vel[i], 0.0, dimX, dimY);
	  ClearArray<<<grid,threads>>>(vel_prev[i], 0.0, dimX, dimY);
  }

  ClearArray<<<grid,threads>>>(chemA, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemA_prev, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(boundary, 0.0, dimX, dimY);

  ClearArray<<<grid,threads>>>(pressure, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(pressure_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(divergence, 0.0, dimX, dimY);

  printf("initGPUArrays(): Initialized GPU arrays.\n");
}

void reset() {
  initGPUArrays();
}

void getMappedPointer(float4 *data, hipGraphicsResource_t cudaGraphRsrc){
  size_t  sizeT;
  hipGraphicsMapResources( 1, &cudaGraphRsrc, 0 );
  hipGraphicsResourceGetMappedPointer((void**)&data, &sizeT, cudaGraphRsrc);
  hipGraphicsUnmapResources( 1, &cudaGraphRsrc, 0 );

}

void drawSquare(float *field, float value) {
  // ClearArray<<<grid,threads>>>(field, 1.0, dimX, dimY);
  DrawSquare<<<grid,threads>>>(field, value, dimX, dimY);
}

void makeColor(float *data, float4 *toDisplay){
  MakeColor<<<grid,threads>>>(data, toDisplay, dimX, dimY);
}
